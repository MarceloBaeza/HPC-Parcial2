#include <hip/hip_runtime.h>
#include <stdio.h>

// Multiplicacion de Fila - Matriz

__global__ void multMatCUDA(double *d_a, double *d_b, double *d_c, int NumberRows_A, int NumberCols_A, int NumberCols_B)
{
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < NumberRows_A && col < NumberCols_B)
  {
    double result = 0;
    for (int k = 0; k < NumberCols_A; k++)
    {
      result += d_a[row * NumberCols_A + k] * d_b[k * NumberCols_B + col];
    }
    d_c[row * NumberCols_B + col] = result;
  }
}

void Mult_Matrix_Cuda(double *h_a, double *h_b, double *h_c, int NumberRows_A, int NumberCols_A, int NumberCols_B)
{
  int blocksize = 32;
  double *d_a, *d_b, *d_c;

  // Asign memory in the device
  hipMalloc(&d_a, sizeof(double) * NumberRows_A * NumberCols_A);
  hipMalloc(&d_b, sizeof(double) * NumberCols_A * NumberCols_B);
  hipMalloc(&d_c, sizeof(double) * NumberRows_A * NumberCols_B);

  hipMemcpy(d_a, h_a, NumberRows_A * NumberCols_A * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, NumberCols_A * NumberCols_B * sizeof(double), hipMemcpyHostToDevice);

  dim3 dimBlock(blocksize, blocksize, 1);
  dim3 dimGrid((NumberCols_B / blocksize) + 1, (NumberRows_A / blocksize) + 1);

  multMatCUDA<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, NumberRows_A, NumberCols_A, NumberCols_B);
  hipMemcpy(h_c, d_c, NumberRows_A * NumberCols_B * sizeof(double), hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}